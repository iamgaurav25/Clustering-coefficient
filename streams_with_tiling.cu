#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>

void cudaCheckError() {                                          
 	hipError_t error=hipGetLastError();                                 
 	if(error!=hipSuccess) {                                              
   		printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));           
   		exit(0); 
 	}                                                                 
}

__global__ void MatrixMulKernel(int *M,int *N,int *P,int SIZE){
	int row = blockIdx.x*blockDim.x+threadIdx.x;
	int col = blockIdx.y*blockDim.y+threadIdx.y;

	if ((row < SIZE) && (col < SIZE)) {
		int output_val = 0;
		for (int k = 0; k < SIZE; ++k) 
			output_val += M[row*SIZE+k]*N[k*SIZE+col];
		P[row*SIZE+col] = output_val;
	}
  
}

__global__ void MatrixMulKernelWithTiling(int *M,int *N,int *P,int Tile_Width,int Width){
	__shared__ double ds_M[16][16];
	__shared__ double ds_N[16][16]; 
	
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int Row=by*blockDim.y+ty;
	int Col=bx*blockDim.x+tx;

    
	int Pvalue=0;
	for(int p=0;p<Width/Tile_Width;p++){
		ds_M[ty][tx]=M[Row*Width+p*Tile_Width+tx];
		ds_N[ty][tx]=N[(p*Tile_Width+ty)*Width+Col];
		__syncthreads();
	for(int i=0;i<Tile_Width;i++)
		Pvalue+=ds_M[ty][i]*ds_N[i][tx];//partial dot product
	__syncthreads();
	}
	P[Row*Width+Col]=Pvalue;//final answer  
}


void print(int* mat, int size){
	for(int i=0;i<size;i++){
		for(int j=0;j<size;j++)
			printf("%d ",mat[i*size+j]);
		printf("\n");
	}
}

int main(int argc, char** argv){
	FILE* fp = fopen(argv[1],"r");
    int *arr1_h,*arr1_d,*arr2_d,*arr3_d,*degree_h, *arr_f1, *arr_f2, *arr_tmp1;
    int a,b;
	int f=0;
    long long int vertices, original_vertices;

	printf("1\n");
	hipStream_t stream1, stream2;
	printf("2\n");
	
	size_t bufferSize = 32;
	size_t offset = bufferSize*bufferSize;
	printf("3\n");
	
	hipStreamCreate(&stream1);
	printf("4\n");
	hipError_t err = hipGetLastError();
	printf("Error: %s\n", hipGetErrorString(err));

	hipStreamCreate(&stream2);
	printf("5\n");
	err = hipGetLastError();
	printf("Error: %s\n", hipGetErrorString(err));

    
	printf("10\n");           
	hipMalloc(&arr1_d,bufferSize*bufferSize*sizeof(int));
	err = hipGetLastError();
	printf("Error: %s\n", hipGetErrorString(err));
	printf("11\n");           
    hipMalloc(&arr2_d,bufferSize*bufferSize*sizeof(int));
	err = hipGetLastError();
	printf("Error: %s\n", hipGetErrorString(err));
	printf("12\n");           
    hipMalloc(&arr3_d,bufferSize*bufferSize*sizeof(int));
	err = hipGetLastError();
	printf("Error: %s\n", hipGetErrorString(err));
	printf("13\n");           
    hipHostAlloc(&arr_tmp1, bufferSize*bufferSize*sizeof(int),hipHostMallocDefault);
	err = hipGetLastError();
	printf("Error: %s\n", hipGetErrorString(err));

	printf("14\n");           
	
	while(fscanf(fp,"%d %d\n",&a,&b)!=EOF){
        if(f==0){
            vertices=a;
            original_vertices=a;
            if(vertices<bufferSize){
                vertices=bufferSize;
            }
            else{
                long long int temp=bufferSize;
                while(vertices>temp){
                    temp=temp*2;
                }
                vertices=temp;
				printf("\nVertices:%lld\n",vertices);
            }

            hipHostAlloc(&arr1_h, vertices*vertices*sizeof(int),hipHostMallocDefault);
            hipHostAlloc(&arr_f1, vertices*vertices*sizeof(int),hipHostMallocDefault);
            hipHostAlloc(&arr_f2, vertices*vertices*sizeof(int),hipHostMallocDefault);
            
			degree_h= (int*)malloc(original_vertices*sizeof(int));

            for (long long int i = 0; i < vertices; ++i)
            {
                degree_h[i]=0;
            }

            for(long long int i=0;i<vertices;i++){
                for(long long int j=0;j<vertices;j++){
                    arr1_h[i*vertices+j]=0;
					arr_f1[i*vertices+j]=0;
                    arr_f2[i*vertices+j]=0;
                       
			   }
            }

            f=1;
        }
        else{

            arr1_h[a*vertices+b]=1;
            arr1_h[b*vertices+a]=1;
            degree_h[a]++;
            degree_h[b]++;
        }
    }
	fclose(fp);

	for(int i=0;i<vertices/bufferSize;i++){
		for(int j=0;j<vertices/bufferSize;j++){
			for(int k=0;k<vertices/bufferSize;k++){
				for(int k1=0;k1<bufferSize;k1++){				
/*					for(int x1=(i*vertices*bufferSize+k1*vertices+k*bufferSize),y1=0;y1<bufferSize;x1++,y1++)
						printf("%d ",arr1_h[x1]	);
					printf("\n");				

					printf("\n");			*/
					hipMemcpyAsync(arr1_d+k1*bufferSize,arr1_h+i*vertices*bufferSize+k1*vertices+k*bufferSize,bufferSize*sizeof(int),hipMemcpyHostToDevice,stream1);
					hipMemcpyAsync(arr2_d+k1*bufferSize,arr1_h+k*vertices*bufferSize+k1*vertices+j*bufferSize,bufferSize*sizeof(int),hipMemcpyHostToDevice,stream2);
//					cudaStreamSynchronize(stream1);
//					cudaStreamSynchronize(stream2);
				}

				hipStreamSynchronize(stream1);
				hipStreamSynchronize(stream2);

				dim3 threadsPerBlock(16, 16);              
				dim3 blocksPerGrid(bufferSize/16, bufferSize/16);       
				MatrixMulKernelWithTiling<<<blocksPerGrid, threadsPerBlock,0, stream1>>>(arr1_d, arr2_d, arr3_d,16,bufferSize); 

				hipStreamSynchronize(stream1);
				hipMemcpyAsync(arr_tmp1,arr3_d,offset*sizeof(int),hipMemcpyDeviceToHost,stream2);
				hipStreamSynchronize(stream2);
				//print(arr_tmp1,bufferSize);
				
				for(int k2=0;k2<bufferSize;k2++)
					for(int k3=0;k3<bufferSize;k3++)
						arr_f1[i*bufferSize*vertices+k2*vertices+j*bufferSize+k3] += arr_tmp1[k2*bufferSize+k3];
						
			}
		}

	}

//	print(arr_f1,vertices);
	
	for(int i=0;i<vertices/bufferSize;i++){
		for(int j=0;j<vertices/bufferSize;j++){
			for(int k=0;k<vertices/bufferSize;k++){
				for(int k1=0;k1<bufferSize;k1++){				
					hipMemcpyAsync(arr1_d+k1*bufferSize,arr_f1+i*vertices*bufferSize+k1*vertices+k*bufferSize,bufferSize*sizeof(int),hipMemcpyHostToDevice,stream1);
					hipMemcpyAsync(arr2_d+k1*bufferSize,arr1_h+k*vertices*bufferSize+k1*vertices+j*bufferSize,bufferSize*sizeof(int),hipMemcpyHostToDevice,stream2);
//					cudaStreamSynchronize(stream1);
//					cudaStreamSynchronize(stream2);
				}

				hipStreamSynchronize(stream1);
				hipStreamSynchronize(stream2);

				dim3 threadsPerBlock(16, 16);              
				dim3 blocksPerGrid(bufferSize/16, bufferSize/16);       
				MatrixMulKernelWithTiling<<<blocksPerGrid, threadsPerBlock,0, stream1>>>(arr1_d, arr2_d, arr3_d,16,bufferSize); 

				hipStreamSynchronize(stream1);
				hipMemcpyAsync(arr_tmp1,arr3_d,offset*sizeof(int),hipMemcpyDeviceToHost,stream2);
				hipStreamSynchronize(stream2);
				//print(arr_tmp1,bufferSize);
				
				for(int k2=0;k2<bufferSize;k2++)
					for(int k3=0;k3<bufferSize;k3++)
						arr_f2[i*bufferSize*vertices+k2*vertices+j*bufferSize+k3] += arr_tmp1[k2*bufferSize+k3];
						
			}
		}

	}
//print(arr_f2,vertices);
				
   	float cc=0;
    for(int i=0;i<original_vertices;i++){
        if(degree_h[i]>=2){
            cc=cc+((float(arr_f2[i*vertices+i]/2))/((degree_h[i]*(degree_h[i]-1))/2));
        }
    }

    cc=cc/original_vertices;
    printf("%f\n",cc);

	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);

	hipHostFree(&arr1_h);
	hipHostFree(&arr_tmp1);
	hipHostFree(&arr1_d);

    hipFree(&arr1_d);
   	hipFree(&arr2_d);
   	hipFree(&arr3_d);

    return 0;
}

