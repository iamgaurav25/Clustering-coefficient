
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;



__global__ void MatrixMulKernel(int *M,int *N,int *P,int Tile_Width,int Width){
	__shared__ double ds_M[32][32];
	__shared__ double ds_N[32][32]; 
	
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int Row=by*blockDim.y+ty;
	int Col=bx*blockDim.x+tx;

    
	int Pvalue=0;
	for(int p=0;p<Width/Tile_Width;p++){
		ds_M[ty][tx]=M[Row*Width+p*Tile_Width+tx];
		ds_N[ty][tx]=N[(p*Tile_Width+ty)*Width+Col];
		__syncthreads();
	for(int i=0;i<Tile_Width;i++)
		Pvalue+=ds_M[ty][i]*ds_N[i][tx];//partial dot product
	__syncthreads();
	}
    
	P[Row*Width+Col]=Pvalue;//final answer
  
}

__global__ void MatrixMulKernel2(int *M,int *N,int *P,int Width){

    int Row=(blockIdx.y*blockDim.y)+threadIdx.y;//row number
    int Col=(blockIdx.x*blockDim.x)+threadIdx.x;//column number
    
    if((Row<Width)&&(Col<Width)){
        float Pvalue=0;
        
        for(int k=0;k<Width;k++){
            Pvalue+=M[Row*Width+k]*N[k*Width+Col];
        }
        P[Row*Width+Col]=Pvalue;//final answer
    }
}

int main(int argc, char** argv){
        ifstream infile(argv[1]);
        int *arr1_h,*arr1_d,*arr2_d,*arr3_d,*degree_h;
        int a,b,bd,gd,tile_width,f=0;
        long long int vertices,edges,size,original_vertices;
        bd=32;
        tile_width=bd;
        while(infile >> a >> b){
        if(f==0){
            vertices=a;
            original_vertices=a;
            edges=b;
            if(vertices<tile_width){
                vertices=tile_width;
            }
            else{
                long long int temp=tile_width;
                while(vertices>temp){
                    temp=temp*2;
                }
                vertices=temp;
            }
            size=(vertices*vertices)*sizeof(int);
            arr1_h=new int[vertices*vertices];
            degree_h=new int[vertices];
            for (long long int i = 0; i < vertices; ++i)
            {
                degree_h[i]=0;
            }
            for(long long int i=0;i<vertices;i++){
                for(long long int j=0;j<vertices;j++){
                    arr1_h[i*vertices+j]=0;
                }
            }
            f=1;
        }
        else{
            arr1_h[a*vertices+b]=1;
            arr1_h[b*vertices+a]=1;
            degree_h[a]++;
            degree_h[b]++;
        }
    }
        hipMalloc(&arr1_d,size);
        hipMalloc(&arr2_d,size);
        hipMalloc(&arr3_d,size);
        hipMemcpy(arr1_d,arr1_h,size,hipMemcpyHostToDevice);

            gd=vertices/bd;
            dim3 grid(gd,gd);
            dim3 block(bd,bd);
            MatrixMulKernel<<< grid,block >>>(arr1_d,arr1_d,arr2_d,tile_width,vertices);

            MatrixMulKernel<<< grid,block >>>(arr2_d,arr1_d,arr3_d,tile_width,vertices);
        
        
        hipMemcpy(arr1_h,arr3_d,size,hipMemcpyDeviceToHost);
        float cc=0;
        for(int i=0;i<vertices;i++){
        if(degree_h[i]>=2){
            cc=cc+((float(arr1_h[i*vertices+i]/2))/((degree_h[i]*(degree_h[i]-1))/2));
        }
    }
    cc=cc/original_vertices;
    cout<<cc<<endl;
    

    return 0;
}
