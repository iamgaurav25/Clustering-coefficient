#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>


void cudaCheckError() {                                          
 	hipError_t error=hipGetLastError();                                 
 	if(error!=hipSuccess) {                                              
   		printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));           
   		exit(0); 
 	}                                                                 
}

__global__ void MatrixMulKernel(int *M,int *N,int *P,int SIZE){
	int row = blockIdx.x*blockDim.x+threadIdx.x;
	int col = blockIdx.y*blockDim.y+threadIdx.y;

	if ((row < SIZE) && (col < SIZE)) {
		double output_val = 0;
		for (int k = 0; k < SIZE; ++k) 
			output_val += M[row*SIZE+k]*N[k*SIZE+col];
		P[row*SIZE+col] = output_val;
	}
  
}

void print(int* mat, int size){
	for(int i=0;i<size;i++){
		for(int j=0;j<size;j++)
			printf("%d ",mat[i*size+j]);
		printf("\n");
	}
}

int main(int argc, char** argv){
	FILE* fp = fopen(argv[1],"r");
    int *arr1_h,*arr1_d,*arr2_d,*arr3_d,*degree_h, *arr_f1, *arr_tmp1;
    int a,b,block_size=16,f=0;
    long long int vertices, original_vertices;

    while(fscanf(fp,"%d %d\n",&a,&b)!=EOF){
        if(f==0){
            vertices=a;
            original_vertices=a;
            if(vertices<block_size){
                vertices=block_size;
            }
            else{
                long long int temp=block_size;
                while(vertices>temp){
                    temp=temp*2;
                }
                vertices=temp;
            }

            hipHostAlloc(&arr1_h, vertices*vertices*sizeof(int),hipHostMallocDefault);
            hipHostAlloc(&arr_f1, vertices*vertices*sizeof(int),hipHostMallocDefault);
            
			degree_h= (int*)malloc(original_vertices*sizeof(int));

            for (long long int i = 0; i < vertices; ++i)
            {
                degree_h[i]=0;
            }

            for(long long int i=0;i<vertices;i++){
                for(long long int j=0;j<vertices;j++){
                    arr1_h[i*vertices+j]=0;
                }
            }

			for(long long int i=0;i<vertices;i++){
                for(long long int j=0;j<vertices;j++){
                    arr_f1[i*vertices+j]=0;
                }
            }
            f=1;
        }
        else{

            arr1_h[a*vertices+b]=1;
            arr1_h[b*vertices+a]=1;
            degree_h[a]++;
            degree_h[b]++;
        }
    }

	print(arr1_h,vertices);

	hipStream_t stream1, stream2;

	size_t bufferSize = 16;
	size_t offset = bufferSize*bufferSize;

	hipStreamCreate(&stream1);

	hipStreamCreate(&stream2);

	hipMalloc(&arr1_d,bufferSize*bufferSize*sizeof(int));
    hipMalloc(&arr2_d,bufferSize*bufferSize*sizeof(int));
    hipMalloc(&arr3_d,bufferSize*bufferSize*sizeof(int));
    hipHostAlloc(&arr_tmp1, bufferSize*bufferSize*sizeof(int),hipHostMallocDefault);
           
	for(int i=0;i<vertices/bufferSize;i++){
		for(int j=0;j<vertices/bufferSize;j++){
			for(int k=0;k<vertices/bufferSize;k++){
				for(int k1=0;k1<bufferSize;k1++){				
					hipMemcpyAsync(arr1_d+k1*bufferSize,arr1_h+(i+k1)*vertices*bufferSize+k*bufferSize,bufferSize*sizeof(int),hipMemcpyHostToDevice,stream1);
					hipMemcpyAsync(arr2_d+k1*bufferSize,arr1_h+(k+k1)*vertices*bufferSize+j*bufferSize,bufferSize*sizeof(int),hipMemcpyHostToDevice,stream2);
				}

				hipStreamSynchronize(stream1);
				hipStreamSynchronize(stream2);

				dim3 threadsPerBlock(16, 16);              
				dim3 blocksPerGrid(bufferSize/16, bufferSize/16);       
				MatrixMulKernel<<<blocksPerGrid, threadsPerBlock,0, stream1>>>(arr1_d, arr2_d, arr3_d,bufferSize); 

				hipStreamSynchronize(stream1);
				hipMemcpyAsync(arr_tmp1,arr3_d,offset*sizeof(int),hipMemcpyDeviceToHost,stream2);

				hipStreamSynchronize(stream2);

				for(int k2=0;k2<bufferSize;k2++)
					for(int k3=0;k3<bufferSize;k3++)
						arr_f1[(i+k2)*bufferSize*vertices+(j*bufferSize+k3)] += arr_tmp1[k2*bufferSize+k3];
			}
		}

	}
    
	for(int i=0;i<vertices/bufferSize;i++){
		for(int j=0;j<vertices/bufferSize;j++){
			for(int k=0;k<vertices/bufferSize;k++){
				for(int k1=0;k1<bufferSize;k1++){				

					hipMemcpyAsync(arr1_d+k1*bufferSize,arr_f1+(i+k1)*vertices*bufferSize+k*bufferSize,bufferSize*sizeof(int),hipMemcpyHostToDevice,stream1);
					hipMemcpyAsync(arr2_d+k1*bufferSize,arr1_h+(k+k1)*vertices*bufferSize+j*bufferSize,bufferSize*sizeof(int),hipMemcpyHostToDevice,stream2);
				}

				hipStreamSynchronize(stream1);
				hipStreamSynchronize(stream2);

				dim3 threadsPerBlock(16, 16);              
				dim3 blocksPerGrid(bufferSize/16, bufferSize/16);       
				MatrixMulKernel<<<blocksPerGrid, threadsPerBlock,0, stream1>>>(arr1_d, arr2_d, arr3_d,bufferSize); 

				hipStreamSynchronize(stream1);
				hipMemcpyAsync(arr_tmp1,arr3_d,offset*sizeof(int),hipMemcpyDeviceToHost,stream2);

				hipStreamSynchronize(stream2);

				for(int k2=0;k2<bufferSize;k2++)
					for(int k3=0;k3<bufferSize;k3++)
						arr_f1[(i+k2)*bufferSize*vertices+(j*bufferSize+k3)] += arr_tmp1[k2*bufferSize+k3];
			}
		}

	}

   	float cc=0;
    for(int i=0;i<original_vertices;i++){
        if(degree_h[i]>=2){
            cc=cc+((float(arr_f1[i*vertices+i]/2))/((degree_h[i]*(degree_h[i]-1))/2));
        }
    }

    cc=cc/original_vertices;
    printf("%f\n",cc);

	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);

	hipHostFree(&arr1_h);
	hipHostFree(&arr_tmp1);
	hipHostFree(&arr1_d);

    hipFree(&arr1_d);
   	hipFree(&arr2_d);
   	hipFree(&arr3_d);

    return 0;
}

