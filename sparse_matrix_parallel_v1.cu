
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

bool sortcol( const int* v1, 
               const int* v2 ) { 
               /*if(v1[1]>v2[1]){
               printf("Pakda gaya\n");
               }*/
 return v1[1] < v2[1]; 
} 

void my_custom_sort(int * &input1,int size){

	int **list1;
	list1=new int*[size];
	for(int i=0;i<size;i++){
		list1[i]=new int[3];
		list1[i][0]=input1[i*3+0];
		list1[i][1]=input1[i*3+1];
		list1[i][2]=input1[i*3+2];
	}
	sort(list1, list1+size,sortcol);
	for(int i=0;i<size;i++){
		input1[i*3+0]=list1[i][0];
		input1[i*3+1]=list1[i][1];
		input1[i*3+2]=list1[i][2];
	}

	for( int i = 0 ; i < size ; i++ )
	{
    	delete[] list1[i]; // delete array within matrix
	}

	delete[] list1;

	}

bool sortcol1( const int* v1, 
               const int* v2 ) { 
 return v1[0] < v2[0]; 
}




void my_custom_sort2(int * &input1,int size){

	
	int **list1;
	list1=new int*[size];
	//printf("cs2\n");
	for(int i=0;i<size;i++){
		list1[i]=new int[3];
	}
	
	for(int i=0;i<size;i++){
		list1[i][0]=input1[i*3+0];
		list1[i][1]=input1[i*3+1];
		list1[i][2]=input1[i*3+2];
	}
	
	sort(list1, list1+size,sortcol1);
	//printf("cs2\n");
	for(int i=0;i<size;i++){
		input1[i*3+0]=list1[i][0];
		input1[i*3+1]=list1[i][1];
		input1[i*3+2]=list1[i][2];
	}
	
	for( int i = 0 ; i < size ; i++ )
	{
    	delete[] list1[i]; // delete array within matrix
	}

	delete[] list1;
	
	}

void transpose(int* input, int size){
	for(int i=0;i<size;i++){
		int tmp = input[i*3+0];
		input[i*3+0] = input[i*3+1];
		input[i*3+1] = tmp;
	}
}

__global__ void transpose_parallel(int * input,int size){

	int tx=(blockIdx.x*256)+threadIdx.x;
	if((tx*3+1)<size){
	int temp=input[tx*3+0];
	input[tx*3+0] = input[tx*3+1];
	input[tx*3+1] = temp;
	}

}

void copy(int* dest, int* src, int size){
	for(int i=0;i<size;i++)
		dest[i] = src[i];
}


void insert(int* output, int row, int col, int val, int &size){
	for(int i=0;i<size;i++)
		if(output[i*3+0]==row && output[i*3+1]==col){
			output[i*3+2]+=val;
			return;
		}
	//output[size] = new int[3];	
	output[size*3+0] = row;
	output[size*3+1] = col;
	output[size*3+2] = val;
	size++;
}

__global__ void oddeven(int* x,int I,int n)
{
	int id=(blockIdx.x*256)+threadIdx.x;
	if(I==0 && ((id*6+4)< n)){
		if(x[id*6+1]>x[id*6+4]){
			int X=x[id*6+1];
			x[id*6+1]=x[id*6+4];
			x[id*6+4]=X;

			X=x[id*6];
			x[id*6]=x[id*6+3];
			x[id*6+3]=X;

			X=x[id*6+2];
			x[id*6+2]=x[id*6+5];
			x[id*6+5]=X;
		}
	}
	if(I==1 && ((id*6+7)< n)){
		if(x[id*6+4]>x[id*6+7]){
			int X=x[id*6+4];
			x[id*6+4]=x[id*6+7];
			x[id*6+7]=X;

			X=x[id*6+3];
			x[id*6+3]=x[id*6+6];
			x[id*6+6]=X;

			X=x[id*6+5];
			x[id*6+5]=x[id*6+8];
			x[id*6+8]=X;
		}
	}
}

__global__ void oddeven2(int* x,int I,int n)
{
	int id=(blockIdx.x*256)+threadIdx.x;
	if(I==0 && ((id*6+3)< n)){
		if(x[id*6]>x[id*6+3]){
			int X=x[id*6+1];
			x[id*6+1]=x[id*6+4];
			x[id*6+4]=X;

			X=x[id*6];
			x[id*6]=x[id*6+3];
			x[id*6+3]=X;

			X=x[id*6+2];
			x[id*6+2]=x[id*6+5];
			x[id*6+5]=X;
		}
	}
	if(I==1 && ((id*6+6)< n)){
		if(x[id*6+3]>x[id*6+6]){
			int X=x[id*6+4];
			x[id*6+4]=x[id*6+7];
			x[id*6+7]=X;

			X=x[id*6+3];
			x[id*6+3]=x[id*6+6];
			x[id*6+6]=X;

			X=x[id*6+5];
			x[id*6+5]=x[id*6+8];
			x[id*6+8]=X;
		}
	}
}

void multiply2(int* &input1, int* &input2, int* &output,int &size1, int &size2, int &size3, int vertices){
	
	int *list1_d,*list2_d;
	//my_custom_sort2(input1, size1);
	hipMalloc(&list1_d,size1*3*sizeof(int));
	hipMemcpy(list1_d,input1,size1*3*sizeof(int),hipMemcpyHostToDevice);
	int gd1,gd2,bd1,bd2;
	gd1=size1/(2*256);
	gd1=gd1+1;
	bd1=256;
	gd2=size2/(2*256);
	gd2=gd2+1;
	bd2=256;
	
	for(int i=0;i<size1;i++){

		//int size=n/2;

		oddeven2<<<gd1,bd1>>>(list1_d,i%2,size1*3);
		
		
	}

	//my_custom_sort2(input2, size2);
	hipMalloc(&list2_d,size2*3*sizeof(int));
	hipMemcpy(list2_d,input2,size2*3*sizeof(int),hipMemcpyHostToDevice);
	for(int i=0;i<size2;i++){

		//int size=n/2;

		oddeven2<<<gd2,bd2>>>(list2_d,i%2,size2*3);
		/*cudaError_t err = cudaGetLastError();
	if (err != cudaSuccess) 
    printf("Error: %s\n", cudaGetErrorString(err));*/
		
	}

	hipMemcpy(input1,list1_d,size1*3*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(input2,list2_d,size2*3*sizeof(int),hipMemcpyDeviceToHost);
	
	output = new int[vertices];
	int outputSize=0;
	int kOld=0;
	int kNew=0;		
	for(int i=0;i<size1;i++){
		int ans=0;		
		int row1= input1[i*3+0];
		int col1= input1[i*3+1];
		int j;
		if(i!=0 && row1==input1[(i-1)*3+0])
			j=kOld;
		else{
			j=kNew;
			kOld = kNew; 		
		}
		while(j<size2){
			int row2 = input2[j*3+0];
			int col2 = input2[j*3+1];			
			if(row1==row2 && col1==col2){				
				insert(output, row1, row2, input1[i*3+2]*input2[j*3+2], outputSize);
				j++;
			}
			else if(row1==row2 && col1!=col2)
				j++;
			else
				break;
		}
		kNew=j;
	}
	
	size3 = outputSize;	
}

void multiply1(int* &input1, int* &input2, int* &output,int* &list1_d ,int &size1, int &size2, int &size3, int vertices){
	
	//transpose(input2, size2);
	int bd,gd;
	if(size1>256){
		gd=size1/256;
		gd=gd+1;
	}
	else{
		gd=1;
		bd=size1;
	}
	hipMalloc(&list1_d,size1*3*sizeof(int));
	hipMemcpy(list1_d,input2,size1*3*sizeof(int),hipMemcpyHostToDevice);
	transpose_parallel<<< gd,bd >>> (list1_d,size1*3);
	//cudaMemcpy(input2,list1_d,size1*3*sizeof(int),cudaMemcpyDeviceToHost);

	
	//my_custom_sort(input2,size2);
	gd=size1/(2*256);
	gd=gd+1;
	bd=256;

	for(int i=0;i<size1;i++){

		//int size=n/2;

		oddeven<<<gd,bd>>>(list1_d,i%2,size1*3);
		
	}
	printf("Final Ans: ");

	
	hipMemcpy(input2,list1_d,size1*3*sizeof(int),hipMemcpyDeviceToHost);
	/*for(int i=0;i<size1;i++){
	printf("%d\n",input2[i*3+1]);
	}*/
	//my_custom_sort(input2,size2);

	copy(input1, input2,size1*3);
	output = new int[3*vertices*vertices];
	int outputSize=0;
	int kOld=0;
	int kNew=0;		
	for(int i=0;i<size1;i++){
		int ans=0;		
		int row1= input1[i*3+0];
		int col1= input1[i*3+1];
		int j;
		if(i!=0 && col1==input1[(i-1)*3+1])
			j=kOld;
		else{
			j=kNew;
			kOld = kNew; 		
		}
		while(j<size2){
			int row2 = input2[j*3+0];
			int col2 = input2[j*3+1];			
			if(col1==col2){				
				insert(output, row1, row2, input1[i*3+2]*input2[j*3+2], outputSize);
				j++;
			}
			else
				break;
		}
		kNew=j;
	}
	size3 = outputSize;	
	multiply2(output, input2, input1, size3, size2, size1, vertices);	
}




int main(int argc, char** argv){


	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	ifstream infile(argv[1]);
	int *list1,*list2,*list3,*list1_d;
	int* degree;	
	int size1, size2, size3;
	int a,b,vertices,edges,f=0, index1=0, index2=0;
	while(infile>>a>>b){
		if(f==0){
			vertices=a;
			edges=b;
			list1=new int[6*edges];
			list2=new int[6*edges];
			degree=new int[vertices];
			for(int i=0;i<vertices;i++)
				degree[i] = 0;
			f=1;
		}
		else{
			list1[index1*3+0]=a;
			list1[index1*3+1]=b;			
			list1[index1*3+2]=1;
			index1++;
			list1[index1*3+0]=b;
			list1[index1*3+1]=a;			
			list1[index1*3+2]=1;
			index1++;
			list2[index2*3+0]=a;
			list2[index2*3+1]=b;			
			list2[index2*3+2]=1;
			index2++;
			list2[index2*3+0]=b;
			list2[index2*3+1]=a;			
			list2[index2*3+2]=1;
			index2++;
			degree[a]++;
			degree[b]++;
		}
	}
	size1= size2=2*edges;
	//auto start = high_resolution_clock::now();
	float milliseconds;
	hipEventRecord(start);
	multiply1(list1,list2, list3,list1_d, size1, size2, size3, vertices);
	hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds,start,stop);


	//multiply2(list3,list2, list1, size3, size2, size1, vertices);	
	//auto stop = high_resolution_clock::now();
	//auto duration = duration_cast<milliseconds>(stop - start);

	float cc=0;
		
	for(int i=0;i<size1;i++){
		if(list1[i*3+0]==list1[i*3+1]){
			if(degree[list1[i*3+0]]>=2){				
				cc=cc+((float(list1[i*3+2]/2))/((degree[list1[i*3+0]]*(degree[list1[i*3+0]]-1))/2));
			}
		}		
	}
	//printf("sdfg\n");
	
	cc=cc/vertices;
	printf("%f\n",cc);
	printf("%f\n",milliseconds);
	//cout << duration.count() << endl;

	return 0;

}
